#include "hip/hip_runtime.h"
/*
 * ELEC 374 - Digital Systems Engineering
 * Machine Problem 1 - Part 1: Device Query
 * Student Name: Sid Prabaharan
 * Student ID: 20351244
 *
 */

#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

int main() {
    // Get number of CUDA devices
    int deviceCount = 0;
    hipError_t error = hipGetDeviceCount(&deviceCount);

    if (error != hipSuccess) {
        printf(" Failed to get device count - %s\n", hipGetErrorString(error));
        return 1;
    }

    // Check if any CUDA devices were found
    if (deviceCount == 0) {
        printf("No CUDA capable devices found\n");
        return 1;
    }

    printf("Found %d CUDA devices\n\n", deviceCount);

    // Iterate through all detected devices
    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);

        printf("Device %d: \"%s\"\n", i, deviceProp.name);
        printf("--------------------------------------------------\n");

        // Get the device clock rate in MHz
        printf("Clock rate: %.2f MHz\n", deviceProp.clockRate * 1e-3f);

        // Number of streaming multiprocessors (SMs)
        printf("Number of streaming multiprocessors (SMs): %d\n", deviceProp.multiProcessorCount);

        // Calculate cores based on compute capability and SM count
        int coresPerSM = 0;
        if (deviceProp.major == 3) {      
            coresPerSM = 192;
        }
        else if (deviceProp.major == 5) { 
            coresPerSM = 128;
        }
        else if (deviceProp.major == 6) { 
            coresPerSM = 64;
            if (deviceProp.minor == 1) coresPerSM = 128;
        }
        else if (deviceProp.major == 7) { 
            if (deviceProp.minor == 0) coresPerSM = 64;
            else coresPerSM = 64;
        }
        else if (deviceProp.major == 8) { 
            coresPerSM = 64;
            if (deviceProp.minor == 6) coresPerSM = 128; 
        }
        else if (deviceProp.major == 9) { 
            coresPerSM = 128;
        }
        else {
            coresPerSM = 32; 
        }

        int totalCores = coresPerSM * deviceProp.multiProcessorCount;
        printf("Number of CUDA cores: %d\n", totalCores);

        // Warp size
        printf("Warp size: %d\n", deviceProp.warpSize);

        // Memory information
        printf("Global memory: %.2f GB\n", deviceProp.totalGlobalMem / (1024.0 * 1024.0 * 1024.0));
        printf("Constant memory: %.2f KB\n", deviceProp.totalConstMem / 1024.0);
        printf("Shared memory per block: %.2f KB\n", deviceProp.sharedMemPerBlock / 1024.0);

        // Registers
        printf("Number of registers available per block: %d\n", deviceProp.regsPerBlock);

        // Thread information
        printf("Maximum number of threads per block: %d\n", deviceProp.maxThreadsPerBlock);

        // Maximum dimensions
        printf("Maximum size of each dimension of a block: [%d, %d, %d]\n",
            deviceProp.maxThreadsDim[0],
            deviceProp.maxThreadsDim[1],
            deviceProp.maxThreadsDim[2]);

        printf("Maximum size of each dimension of a grid: [%d, %d, %d]\n",
            deviceProp.maxGridSize[0],
            deviceProp.maxGridSize[1],
            deviceProp.maxGridSize[2]);

        printf("\n");
    }

    return 0;
}
